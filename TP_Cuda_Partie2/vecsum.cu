
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024

/*
Le principe :

On n'a pas assez de threads sur un bloc pour s'occuper de tous les éléments du tableau.
Donc on va "découper" ce grand tableau en plusieurs plus petits.
Chaque "sous-tableau" va faire le calcul de sa somme. On récupère ensuite le résultat de la somme des "sous tableau"
Qu'on place dans un tableau temporaire. Et on calcule ensuite la somme de ce tableau temporaire.

Je mets "sous tableau" entre guillemets car on ne crée pas de sous tableau. cf vidéo du cours qui expliquera mieux que moi.
*/


void reduce(unsigned int *vec,unsigned int*sum, int size);
__device__ void kreduceBlock (unsigned int *d_vec, int size);
__global__ void kreduce2 (unsigned int *d_tmp, int size);
__global__ void kreduce1 (unsigned int *d_in, unsigned int *d_tmp, int size );

int main(int argc, char **argv){
  if (argc < 2){
     printf("Usage: <filename>\n");
     exit(-1);
   }
   int size;
   unsigned int *vec;
   FILE *f = fopen(argv[1],"r");
   fscanf(f,"%d\n",&size);
   size = 1 << size;
   if (size >= (1 << 20)){
     printf("Size (%u) is too large: size is limited to 2^20\n",size);
     exit(-1);
   }
    vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
   for (int i=0; i<size; i++){
     fscanf(f, "%u\n",&(vec[i]));
   }
   unsigned int sum=0;


   reduce(vec,&sum,size);
   printf("sum = %u\n", sum);

  unsigned int sum2 = 0;
  for (int i=0; i<size; i++){
    sum2 += vec[i];
  }
  printf("sum2 = %u\n", sum2);

  fclose(f);
  return 0;
}

void reduce(unsigned int *vec,unsigned int *sum, int size){

  unsigned int *d_vec,*d_tmp;
  int bytes = size*sizeof(unsigned int);
  hipMalloc((void **)&d_vec, bytes);
  hipMalloc((void **)&d_tmp, bytes /BLOCK_SIZE +1);
  hipMemcpy(d_vec,vec,bytes,hipMemcpyHostToDevice);
  int num_blocks = size / BLOCK_SIZE;
  if (size % BLOCK_SIZE) num_blocks ++;

  kreduce1<<<num_blocks,BLOCK_SIZE>>>(d_vec,d_tmp,size);

  /*Après kreduce1 notre d_tmp est donc remplie des sommes des "sous tableau" de d_vec
  On s'est donc ramené à un tableau d'une taille < 1024 qui peut être traité par un seul block
*/
  kreduce2<<<1,size/BLOCK_SIZE +1>>>(d_tmp,size/BLOCK_SIZE +1);
  

  hipMemcpy(sum,d_tmp,sizeof(unsigned int),hipMemcpyDeviceToHost);
  hipFree(d_vec);hipFree(d_tmp);
}

__device__ void kreduceBlock (unsigned int *d_vec, int size){
  int id=threadIdx.x;
  for(int i=size/2;i>=1;i>>=1){
    if(id<i && id+i<size){
      d_vec[id]+=d_vec[id+i];}
      __syncthreads();

  }}

  __global__ void kreduce1 (unsigned int *d_vec, unsigned int *d_tmp, int size ){
    int block_id = blockIdx.x;
    int offset = block_id * BLOCK_SIZE;
    int n = BLOCK_SIZE;
    if ( block_id == gridDim.x-1){// dernier bloc
      n = size-block_id*BLOCK_SIZE;
    }
    kreduceBlock(&(d_vec[offset]),n);
    /*On place de résultat obtenu dans le "sous tableau" dans le 
    tableau temporaire a l'indice du block.
    Un thread par block s'occupe de cette action(car une seule action a faire) */
    if (threadIdx.x== 0){
      d_tmp[block_id] = d_vec[offset] ;
    }
  }

    __global__ void kreduce2 (unsigned int *d_tmp, int size){
      int id=threadIdx.x;
      for(int i=size/2;i>=1;i>>=1){
        if(id<i && id+i<size){
          d_tmp[id]+=d_tmp[id+i];}
          __syncthreads();

      }}
