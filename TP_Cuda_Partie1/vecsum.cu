
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

//Déclaration des prototypes des fonctions du fichier.
void reduce(unsigned int *vec,unsigned int*sum, int size);
__global__ void kreduce (unsigned int *tab, int size);

/*
L'intérieur du main reprend le code de vecsum.c (la version en code séquentiel)

On va récupérer l'argument qu'on a donné quand on exécute le ficher compilé.
Cet argument correspond au fichier vecteur.txt

On va ensuite le charger dans la mémoire à l'adresse du pointeur vec

*/
int main(int argc, char **argv){
  
  if (argc < 2){
    //Argument non valide
     printf("Usage: <filename>\n");
     exit(-1);
   }
   int size;
   unsigned int *vec;

   //On ouvre le fichier qu'on "place" dans f 
   FILE *f = fopen(argv[1],"r");
  //On récupère sa taille
   fscanf(f,"%d\n",&size);

   size = 1 << size;
   if (size >= (1 << 20)){
     printf("Size (%u) is too large: size is limited to 2^20\n",size);
     exit(-1);
   }
  
  //allocation de l'espace nécessaire au tableau vec
  vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);

  //boucle qui lit tous les éléments du fichier et les met dans vec
   for (int i=0; i<size; i++){
     fscanf(f, "%u\n",&(vec[i]));
   }
   unsigned int sum=0;

  /*Appel de la fonction de base avec en paramètre : 
  le vecteur
  l'addresse où doit être stocker la somme
  la taille du vecteur

  On donne l'adresse du sum (en utilisant "&" avant) pour pouvoir modifier sa valeur en s'addressant 
  directement a l'espace mémoire où est stocker sum et non à la variable sum
  */
  reduce(vec,&sum,size);
  printf("sum = %u\n", sum);


  /*
  Code séquentiel basique qui fait la somme du vecteur de façon classique (permet de vérifier)
  unsigned int sum2 = 0;
  for (int i=0; i<size; i++){
    sum2 += vec[i];
  }
  printf("sum2 = %u\n", sum2);
  */
  fclose(f);
  return 0;
}

void reduce(unsigned int *vec,unsigned int *sum, int size){

  unsigned int *d_vec;
  int bytes = size*sizeof(unsigned int);
  //Allocation de l'espace mémoire du gpu pour stocker le tableau vecteur
  hipMalloc((void **)&d_vec, bytes);
  //Copie des donnec dans le gpu
  hipMemcpy(d_vec,vec,bytes,hipMemcpyHostToDevice);

  kreduce<<<1,size>>>(d_vec,size);

  /*Copie du résultat :
  On fait un Memcpy de d_vec : d_vec est un tableau mais comme on met juste d_vec sans indice (d_vec[x])
  alors d_vec vaut le premier élément du tableau => d_vec = d_vec[0] 
  */
  hipMemcpy(sum,d_vec,sizeof(unsigned int),hipMemcpyDeviceToHost);
  hipFree(d_vec);
}

__global__ void kreduce (unsigned int *tab, int size){
  int id=threadIdx.x;
  for(int i=size/2;i>0;i=i/2){
    if(id<i && id+i<size){
      tab[id]+=tab[id+i];}
      __syncthreads();

  }}
